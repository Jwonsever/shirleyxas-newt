
#include <hip/hip_runtime.h>
#if defined ( __PW_CUDA )

#include <cuda_runtime.h>
#include <cufft.h>
#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>

/*extern void cuda_error_check (cudaError_t cudaError) {
  if (cudaError != cudaSuccess) {
    printf("CUDA RT Error: %s\n", cudaGetErrorString(cudaError));
    fflush(stdout);
    exit(-1);
  }
}*/

extern void cuda_error_check2 (cudaError_t cudaError, int line) {
  int         pid;
  size_t      free, total;
  cudaError_t cErr2;

  cErr2 = cudaGetLastError();
  if (cudaError != cudaSuccess || cErr2 != cudaSuccess) {
    pid = getpid();
    printf("%d CUDA RT Error line %d\n", pid, line);
    printf("%d CUDA RT1 Error: %s\n", pid, cudaGetErrorString(cudaError));
    printf("%d CUDA RT2 Error: %s\n", pid, cudaGetErrorString(cErr2));
    cudaMemGetInfo(&free,&total);
    printf("%d Free: %zu , Total: %zu\n", pid, free, total);
    fflush(stdout);
    exit(-1);
  }
}

/*extern void cufft_error_check (cufftResult_t cufftError) {
  if (cufftError != CUFFT_SUCCESS) {
    switch (cufftError) {
      case CUFFT_INVALID_PLAN:   printf("CUDA FFT Error (CUFFT_INVALID_PLAN)\n"); break;
      case CUFFT_ALLOC_FAILED:   printf("CUDA FFT Error (CUFFT_ALLOC_FAILED)\n"); break;
      case CUFFT_INVALID_VALUE:  printf("CUDA FFT Error (CUFFT_INVALID_VALUE)\n"); break;
      case CUFFT_INTERNAL_ERROR: printf("CUDA FFT Error (CUFFT_INTERNAL_ERROR)\n"); break;
      case CUFFT_EXEC_FAILED:    printf("CUDA FFT Error (CUFFT_EXEC_FAILED)\n"); break;
      case CUFFT_INVALID_SIZE:   printf("CUDA FFT Error (CUFFT_INVALID_SIZE)\n"); break;
      default: printf("CUDA FFT Error (--unimplemented--)\n"); break;
    }
    fflush(stdout);
    exit(-1);
  }
}*/

extern void cufft_error_check2 (cufftResult_t cufftError, int line) {
  int         pid;
  size_t      free, total;
  cudaError_t cErr2;

  cErr2 = cudaGetLastError();
  if (cufftError != CUFFT_SUCCESS || cErr2 != cudaSuccess) {
    pid = getpid();
    printf("%d CUDA FFT Error line: %d \n", pid, line);
    switch (cufftError) {
      case CUFFT_INVALID_PLAN:   printf("%d CUDA FFT1 Error (CUFFT_INVALID_PLAN)\n", pid); break;
      case CUFFT_ALLOC_FAILED:   printf("%d CUDA FFT1 Error (CUFFT_ALLOC_FAILED)\n", pid); break;
      case CUFFT_INVALID_VALUE:  printf("%d CUDA FFT1 Error (CUFFT_INVALID_VALUE)\n", pid); break;
      case CUFFT_INTERNAL_ERROR: printf("%d CUDA FFT1 Error (CUFFT_INTERNAL_ERROR)\n", pid); break;
      case CUFFT_EXEC_FAILED:    printf("%d CUDA FFT1 Error (CUFFT_EXEC_FAILED)\n", pid); break;
      case CUFFT_INVALID_SIZE:   printf("%d CUDA FFT1 Error (CUFFT_INVALID_SIZE)\n", pid); break;
      default: printf("%d CUDA FFT1 Error (--unimplemented--) %d %d\n", pid, cufftError, cErr2); break;
    }
    printf("%d CUDA FFT2 Error %s \n", pid, cudaGetErrorString(cErr2));
    cudaMemGetInfo(&free,&total);
    printf("%d Free: %zu , Total: %zu\n", pid, free, total);
    fflush(stdout);
    exit(-1);
  }
}

#endif
